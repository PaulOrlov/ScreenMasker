#include "hip/hip_runtime.h"
#include "stdafx.h"
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

unsigned char *patternPtrGpu;
unsigned char *stencilPtrGpu;
unsigned int *backgroundImageGpu;

int blocksQuantity = 0;
int threadsQuantity = 0;

__device__ void computePixelValue(	unsigned int bA1, 
									unsigned int bA2, 
									unsigned int desiredColor, 
									unsigned int *outMass, 
									int massIndex	)
{
	unsigned int bA = (bA1 * bA2) >> 8;

	unsigned int bR = bA * (desiredColor >> 16 & 0xff) >> 8;
	unsigned int bG = bA * (desiredColor >> 8 & 0xff) >> 8;
	unsigned int bB = bA * (desiredColor & 0xff) >> 8;

	outMass[massIndex] = bA<<24 | bR<<16 | bG<<8 | bB;
}

__global__ void drawBackgroundKernel(	unsigned char *patternPtrGpu, 
										unsigned char *stencilPtrGpu, 
										int patternWidth, 
										int patternHeight, 
										int stencilWidth, 
										int stencilHeight, 
										int screenWidth, 
										int screenHeight, 
										int cursorX, 
										int cursorY, 
										unsigned int desiredColor, 
										unsigned int *backgroundImageGpu	)
{
	int threadIndex = blockIdx.x*blockDim.x + threadIdx.x;
	int xCoord = threadIndex%screenWidth;
	int yCoord = threadIndex/screenWidth;

	if(xCoord < screenWidth && yCoord < screenHeight)
	{
		int bgXCoord = xCoord%patternWidth;
		int bgYCoord = yCoord%patternWidth;
		int bgMassIndex = bgYCoord*patternWidth + bgXCoord;

		int x0 = cursorX - stencilWidth*0.5;
		int x1 = cursorX + stencilWidth*0.5;
		int y0 = cursorY - stencilHeight*0.5;
		int y1 = cursorY + stencilHeight*0.5;

		if( xCoord >= x0 && 
			xCoord < x1 && 
			yCoord >= y0 && 
			yCoord < y1 )
		{
			int stencilXCoord = xCoord - x0;
			int stencilYCoord = yCoord - y0;
			int stencilMassIndex = stencilYCoord*stencilWidth + stencilXCoord;

			computePixelValue(((~patternPtrGpu[bgMassIndex]) & 0xff), ((~stencilPtrGpu[stencilMassIndex]) & 0xff), desiredColor, backgroundImageGpu, threadIndex);
		}
		else
		{
			computePixelValue((~patternPtrGpu[bgMassIndex] & 0xff), 0xff, desiredColor, backgroundImageGpu, threadIndex);
		}
	}
}

int initGpuResources(	UINT8 *patternPtr, 
						UINT8 *stencilPtr, 
						int patternWidth, 
						int patternHeight, 
						int stencilWidth, 
						int stencilHeight, 
						int screenWidth, 
						int screenHeight	)
{
	int deviceCount = -1;
	hipGetDeviceCount(&deviceCount);

	if(deviceCount < 1)
	{
		return -1;
	}

	int cudaDevice = 0;
	hipSetDevice(cudaDevice);

	hipDeviceProp_t cudaDeviceProps;

	hipGetDeviceProperties(&cudaDeviceProps, cudaDevice);

	hipMalloc(&backgroundImageGpu, sizeof(unsigned int)*screenWidth*screenHeight);
	hipMalloc(&patternPtrGpu, sizeof(unsigned char)*patternWidth*patternHeight);
	hipMalloc(&stencilPtrGpu, sizeof(unsigned char)*stencilWidth*stencilHeight);

	hipMemcpy(patternPtrGpu, patternPtr, sizeof(unsigned char)*patternWidth*patternHeight, hipMemcpyHostToDevice);
	hipMemcpy(stencilPtrGpu, stencilPtr, sizeof(unsigned char)*stencilWidth*stencilHeight, hipMemcpyHostToDevice);

	threadsQuantity = cudaDeviceProps.maxThreadsPerBlock;
	blocksQuantity = (int)(ceil((float)(screenWidth*screenHeight)/(float)(threadsQuantity)));

	return 0;
}

int drawBackground(	int patternWidth, 
					int patternHeight, 
					int stencilWidth, 
					int stencilHeight, 
					int screenWidth, 
					int screenHeight, 
					int cursorX, 
					int cursorY, 
					UINT32 desiredColor, 
					UINT32 *backgroundImage	)
{
	//hipMemset(backgroundImageGpu, 0, sizeof(unsigned int)*screenWidth*screenHeight);

	drawBackgroundKernel<<<blocksQuantity, threadsQuantity>>>(	patternPtrGpu, 
																stencilPtrGpu, 
																patternWidth, 
																patternHeight, 
																stencilWidth, 
																stencilHeight, 
																screenWidth, 
																screenHeight, 
																cursorX, 
																cursorY, 
																desiredColor, 
																backgroundImageGpu	);

	hipMemcpy(backgroundImage, backgroundImageGpu, sizeof(unsigned int)*screenWidth*screenHeight, hipMemcpyDeviceToHost);

	return 0;
}

int freeGpuResources()
{
	hipFree(patternPtrGpu);
	hipFree(stencilPtrGpu);
	hipFree(backgroundImageGpu);

	return 0;
}